// #define USE_CUBLAS

#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>
#ifdef USE_CUBLAS
#include <cublas_v2.h>
#endif

#include <cmath>
#include <nvtx3/nvToolsExt.h>
using namespace std;

const int TILE_WIDTH = 64;	// 定义块block大小

__global__ void MatrixMulSharedMemKernel_v1(float *A,
    float *B, float *C, int wA,
    int wB) {

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int Crow = bx * TILE_WIDTH + ty;
    int Ccol = by * TILE_WIDTH + tx;
    // 写入(Crow, Ccol)

    // 每次读取一个block的A和B
    // (Arow, Acol)
    int AleftRowPoint = bx * TILE_WIDTH;
    int AleftColPoint = 0;
    int AEndColPoint = wA;
    
    int BleftRowPoint = 0;
    int BleftColPoint = by * TILE_WIDTH;

    float cval = 0.0f;

    for(; AleftColPoint < AEndColPoint; AleftColPoint += TILE_WIDTH, BleftRowPoint += TILE_WIDTH) {
        __shared__ float As[TILE_WIDTH+1][TILE_WIDTH+1];
        __shared__ float Bs[TILE_WIDTH+1][TILE_WIDTH+1];
        if(AleftRowPoint + ty < wA && AleftColPoint + tx < wA) {
            As[ty][tx] = A[(AleftRowPoint + ty) * wA + AleftColPoint + tx];
        } else {
            As[ty][tx] = 0.0f;
        }

       if(BleftRowPoint + ty < wB && BleftColPoint + tx < wB) {
            Bs[ty][tx] = B[(BleftRowPoint + ty) * wB + BleftColPoint + tx];
        } else {
            Bs[ty][tx] = 0.0f;
        }

        __syncthreads();

        #pragma unroll
        for(int k = 0; k < TILE_WIDTH; k++) {
            cval = fma(As[ty][k], Bs[k][tx], cval);
        }

        __syncthreads();
    }

    if(Crow < wA && Ccol < wB) {
       C[Crow * wB + Ccol] = cval;
    }
}

// each thread compute 8 elements
__global__ void MatrixMulSharedMemKernel_v2(float *A,
    float *B, float *C, int wA,
    int wB) {
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x; // 64

    const int TILE_C = 8; 
    int wK = TILE_WIDTH / TILE_C;
    int Ccol = bx * TILE_WIDTH;
    int Crow = by * TILE_WIDTH;
    __shared__ float As[64+1][8+1];
    __shared__ float Bs[8+1][64+1];
    float cval[TILE_C] = {0.0f, 0.0f, 0.0f, 0.0f};

    float *A_Left_Pointer = A + Crow * wA;
    float *B_Left_Pointer = B + Ccol;

    C += (Crow * wB + Ccol);

    int innerColA = tx % wK;
    int innerRowA = tx / wK;
    int innerColB = tx % TILE_WIDTH;
    int innerRowB = tx / TILE_WIDTH;

    int threadCol = tx % TILE_WIDTH;
    int threadRow = tx / TILE_WIDTH;

    for(int tile_idx = 0; tile_idx < wA; tile_idx += TILE_C) {

        As[innerRowA][innerColA] = A_Left_Pointer[innerRowA * wA + innerColA];
        Bs[innerRowB][innerColB] = B_Left_Pointer[innerRowB * wB + innerColB];

        __syncthreads();

        A_Left_Pointer += wK;
        B_Left_Pointer += wK * wB;

        for(int bval_idx = 0; bval_idx < wK; bval_idx++) {
          // 外积         
          float b_tmp = Bs[bval_idx][threadCol];
          for(int a_idx = 0; a_idx < TILE_C; a_idx++) {
            cval[a_idx] = fma(As[threadRow * TILE_C + a_idx][bval_idx], b_tmp, cval[a_idx]);
          }
        }

        __syncthreads();
    }

    for(int i = 0; i < TILE_C; i++) {
      C[(threadRow * TILE_C + i) * wB + threadCol] = cval[i];
    }

}

//! For square matrices only
__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int width)
{
  // 计算行索引
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  
  // 计算列索引
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  
  // 确保线程在矩阵范围内
  if ((row < width) && (col < width)) {
    float pValue = 0.0;
    
    // 计算矩阵乘法的一个元素
    for (int k = 0; k < width; k++) {
      pValue += d_M[row * width + k] * d_N[k * width + col];
    }
    
    // 将计算结果存储到输出矩阵中
    d_P[row * width + col] = pValue;
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set matrix multiply on CPU
//! C = A * B
//! @param C          reference data, computed but preallocated
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param hA         height of matrix A
//! @param wA         width of matrix A
//! @param wB         width of matrix B
////////////////////////////////////////////////////////////////////////////////
void
matrixMulCPU(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j)
        {
            double sum = 0;

            for (unsigned int k = 0; k < wA; ++k)
            {
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }

            C[i * wB + j] = (float)sum;
        }
}

void printDiff(float *data1, float *data2, int width, int height, int iListLength, float fListTol)
{
    printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
    int i,j,k;
    int error_count=0;

    for (j = 0; j < height; j++)
    {
        for (i = 0; i < width; i++)
        {
            k = j * width + i;
            float fDiff = fabs(data1[k] - data2[k]);

            if (fDiff > fListTol)
            {
                if (error_count < iListLength)
                {
                    printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }

                error_count++;
            }
        }
    }

    printf(" \n  Total Errors = %d\n", error_count);
}

void getArg(int argc, char* argv[], int &size, int &check)
{
  if (argc != 3)
  {
    cerr << "Usage: " << argv[0] << " <check_enable> <size>\n";
    cerr << "\tcheck_enable: 1 to enable result checking\n";
    cerr << "\tsize: size of the matrix\n";
    exit(1);
  }

  int val1, val2;
  try
  {
    val1 = stoi(argv[1]);
    val2 = stoi(argv[2]);
  }
  catch (const invalid_argument& e)
  {
    cerr << "ERROR: parameters should be integer\n";
    exit(1);
  }

  check = val1;
  size = val2;
}



int main(int argc, char* argv[])
{
  int size, check;
  getArg(argc, argv, size, check);

  int m = size, n = size, k = size;
  
  // 声明存放在GPU上的数组
  float *h_M, *h_N, *d_M, *d_N;
  float *h_P, *d_P;
  
  size_t sizeM = m * k * sizeof(float);
  size_t sizeN = k * n * sizeof(float);
  size_t sizeP = m * n * sizeof(float);


  // Allocate host memory
  h_M = (float*) malloc(sizeM);
  h_N = (float*) malloc(sizeN);
  h_P = (float*) malloc(sizeP);
  float *reference = (float *)malloc(sizeP);

  // Allocate device memory
  hipMalloc(&d_M, sizeM);
  hipMalloc(&d_N, sizeN);
  hipMalloc(&d_P, sizeP);

  // Init data 
  for(int i = 0; i < m * n; ++i)
  {
    if(i % 2 == 0)
      h_M[i] = 1.0;
    else
      h_M[i] = 0.5;
  }

  for(int i = 0;i < n * k; ++i)
  {
    if(i % 2 == 0)
      h_N[i] = 0.5;
    else
      h_N[i] = 1.0;
  }

  // Copy data from CPU to GPU
  hipMemcpy(d_M, h_M, sizeM, hipMemcpyHostToDevice);
  hipMemcpy(d_N, h_N, sizeN, hipMemcpyHostToDevice);

  #define MatrixMulSharedMemKernel MatrixMulSharedMemKernel_v2

   // Launch kernel 定义grid&block
  dim3 grid((int)ceil(k*1.0 / TILE_WIDTH), (int)ceil(m*1.0/ TILE_WIDTH));
  dim3 block(TILE_WIDTH * TILE_WIDTH / 8);
  
  printf("Grid: (%d, %d)\n", grid.x, grid.y);
  printf("Block(y, x): (%d, %d)\n", block.y, block.x);


    const float alpha = 1.0f;
    const float beta  = 0.0f;
  // 添加 warmup
  {
    nvtxRangePushA("Warmup Start");
    printf("Warming up...\n");
    for (int i = 0; i < 3; i++) {
        MatrixMulSharedMemKernel<<<grid, block>>>(d_M, d_N, d_P, m, n);
    }
    hipDeviceSynchronize();
    nvtxRangePop();
  }

  // Timing records 
    nvtxRangePushA("Kernel Execution Start");
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    int nIter = 5;
#ifdef USE_CUBLAS
    cublasHandle_t handle;
    cublasCreate(&handle);
#endif

    for (int j = 0; j < nIter; j++) {
        //matrixMulCPU(reference, h_M, h_N, m, k, n);
        // MatrixMulKernel<<<grid, block>>>(d_M, d_N, d_P, m);
        MatrixMulSharedMemKernel<<<grid, block>>>(d_M, d_N, d_P, m, n);
        // cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, n, m, k, &alpha, d_N, n, d_M, k, &beta, d_P, n);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    nvtxRangePop();
    float msecPerMatrixMul;
    hipEventElapsedTime(&msecPerMatrixMul, start, stop);
    msecPerMatrixMul /= nIter;
    printf("Kernel Elapsed Time: %.3f ms\n", msecPerMatrixMul);

  // Compute and print the performance
  double flopsPerMatrixMul = 2.0 * (double)m * (double)n * (double)k;
  double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
  printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
		  gigaFlops,
		  msecPerMatrixMul,
		  flopsPerMatrixMul);

  // Copy data from GPU to CPU 
  hipMemcpy(h_P, d_P, sizeP, hipMemcpyDeviceToHost);

  // compute reference solution
  if (check == 1)
  {
    printf("Computing result using host CPU...");
    matrixMulCPU(reference, h_M, h_N, m, k, n);
    printf("done.\n");
    printDiff(reference, h_P, n, m, 100, 1.0e-5f);
  }

  free(h_P);
  free(h_M);
  free(h_N);
  hipFree(d_P);
  hipFree(d_M);
  hipFree(d_N);
#ifdef USE_CUBLAS
  cublasDestroy(handle);
#endif

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      printf("CUDA Error: %s\n", hipGetErrorString(err));
      return -1;
  }

  return 0;
}

